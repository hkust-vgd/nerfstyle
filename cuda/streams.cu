#include "streams.cuh"
#include <iostream>

hipStream_t *StreamPool::streamArray = nullptr;
int64_t StreamPool::counter = 0;
int64_t StreamPool::num_streams = 0;

void StreamPool::init(int64_t num_streams) {
    StreamPool::streamArray = (hipStream_t*)malloc(num_streams * sizeof(hipStream_t*));
    StreamPool::num_streams = num_streams;
    StreamPool::counter = 0;

    for (int i = 0; i < num_streams; i++) {
        hipError_t cudaErr = hipStreamCreate(&StreamPool::streamArray[i]);
        if (cudaErr != hipSuccess)
            std::cerr << "Cannot create stream " << i << std::endl;
    }
}

void StreamPool::destroy() {
    free(streamArray);
}

hipStream_t StreamPool::get_next_stream() {
    hipStream_t cur_stream = streamArray[counter];
    counter = (counter + 1) % num_streams;
    return cur_stream;
}
