#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>

#include <ATen/cuda/HIPContext.h>
#include <torch/torch.h>

#include <cstdio>
#include <stdint.h>
#include <stdexcept>
#include <limits>

#define CHECK_CUDA(x) TORCH_CHECK(x.device().is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be a contiguous tensor")
#define CHECK_IS_INT(x) TORCH_CHECK(x.scalar_type() == at::ScalarType::Int, #x " must be an int tensor")
#define CHECK_IS_FLOATING(x) TORCH_CHECK(x.scalar_type() == at::ScalarType::Float || x.scalar_type() == at::ScalarType::Half || x.scalar_type() == at::ScalarType::Double, #x " must be a floating tensor")


inline constexpr __device__ float SQRT3() { return 1.7320508075688772f; }
inline constexpr __device__ float RSQRT3() { return 0.5773502691896258f; }
inline constexpr __device__ float PI() { return 3.141592653589793f; }
inline constexpr __device__ float RPI() { return 0.3183098861837907f; }


template <typename T>
inline __host__ __device__ T div_round_up(T val, T divisor) {
    return (val + divisor - 1) / divisor;
}

inline __host__ __device__ float signf(const float x) {
    return copysignf(1.0, x);
}

inline __host__ __device__ float clamp(const float x, const float min, const float max) {
    return fminf(max, fmaxf(min, x));
}

inline __host__ __device__ void swapf(float& a, float& b) {
    float c = a; a = b; b = c;
}

inline __device__ int mip_from_pos(const float x, const float y, const float z, const float max_cascade) {
    const float mx = fmaxf(fabsf(x), fmaxf(fabs(y), fabs(z)));
    int exponent;
    frexpf(mx, &exponent); // [0, 0.5) --> -1, [0.5, 1) --> 0, [1, 2) --> 1, [2, 4) --> 2, ...
    return fminf(max_cascade - 1, fmaxf(0, exponent));
}

inline __device__ int mip_from_dt(const float dt, const float H, const float max_cascade) {
    const float mx = dt * H * 0.5;
    int exponent;
    frexpf(mx, &exponent);
    return fminf(max_cascade - 1, fmaxf(0, exponent));
}

inline __host__ __device__ uint32_t __expand_bits(uint32_t v)
{
	v = (v * 0x00010001u) & 0xFF0000FFu;
	v = (v * 0x00000101u) & 0x0F00F00Fu;
	v = (v * 0x00000011u) & 0xC30C30C3u;
	v = (v * 0x00000005u) & 0x49249249u;
	return v;
}

inline __host__ __device__ uint32_t __morton3D(uint32_t x, uint32_t y, uint32_t z)
{
	uint32_t xx = __expand_bits(x);
	uint32_t yy = __expand_bits(y);
	uint32_t zz = __expand_bits(z);
	return xx | (yy << 1) | (zz << 2);
}

inline __host__ __device__ uint32_t __morton3D_invert(uint32_t x)
{
	x = x & 0x49249249;
	x = (x | (x >> 2)) & 0xc30c30c3;
	x = (x | (x >> 4)) & 0x0f00f00f;
	x = (x | (x >> 8)) & 0xff0000ff;
	x = (x | (x >> 16)) & 0x0000ffff;
	return x;
}

// Element-wise operations

inline __host__ __device__ float3 operator+(const float3 &a, const float3 &b) {
    return make_float3(a.x + b.x, a.y + b.y, a.z + b.z);
}

inline __host__ __device__ float3 operator-(const float3 &a, const float3 &b) {
    return make_float3(a.x - b.x, a.y - b.y, a.z - b.z);
}

inline __host__ __device__ float3 operator*(const float3 &a, const float3 &b) {
    return make_float3(a.x * b.x, a.y * b.y, a.z * b.z);
}

inline __host__ __device__ float fminf(const float3 &v) {
    return fminf(v.x, fminf(v.y, v.z));
}

// Scalar operations

inline __host__ __device__ float3 operator+(const float a, const float3 &b) {
    return make_float3(a + b.x, a + b.y, a + b.z);
}

inline __host__ __device__ float3 operator+(const float3 &a, const float b) {
    return b + a;
}

inline __host__ __device__ float3 operator-(const float a, const float3 &b) {
    return make_float3(a - b.x, a - b.y, a - b.z);
}

inline __host__ __device__ float3 operator-(const float3 &a, const float b) {
    return make_float3(a.x - b, a.y - b, a.z - b);
}

inline __host__ __device__ float3 operator*(const float a, const float3 &b) {
    return make_float3(a * b.x, a * b.y, a * b.z);
}

inline __host__ __device__ float3 operator*(const float3 &a, const float b) {
    return b * a;
}

inline __host__ __device__ float3 operator/(const float a, const float3 &b) {
    return make_float3(a / b.x, a / b.y, a / b.z);
}

inline __host__ __device__ float3 operator/(const float3 &a, const float b) {
    return make_float3(a.x / b, a.y / b, a.z / b);
}

// Vector math
inline __host__ __device__ float dot(const float3 &a, const float3 &b) {
    return a.x * b.x + a.y * b.y + a.z * b.z;
}

inline __host__ __device__ float norm(const float3 &v) {
    return sqrtf(dot(v, v));
}

// Assignment
template <typename scalar_t>
inline __host__ __device__ void write_float3(const float3 &v, scalar_t * dest) {
    dest[0] = v.x;
    dest[1] = v.y;
    dest[2] = v.z;
}

// Conversion
inline __host__ __device__ const uint3& to_uint3(const float3& v) {
    return make_uint3((uint)v.x, (uint)v.y, (uint)v.z);
}

inline __host__ __device__ const float3& to_float3(const uint3& v) {
    return make_float3((float)v.x, (float)v.y, (float)v.z);
}

// Other overloads for float3

inline __host__ __device__ float3 clamp(const float3 &v, const float min, const float max) {
    return make_float3(
        fminf(max, fmaxf(min, v.x)),
        fminf(max, fmaxf(min, v.y)),
        fminf(max, fmaxf(min, v.z))
    );
}

inline __device__ int mip_from_pos(const float3 &v, const float max_cascade) {
    return mip_from_pos(v.x, v.y, v.z, max_cascade);
}

inline __host__ __device__ uint32_t __morton3D(uint3 v) {
    return __morton3D(v.x, v.y, v.z);
}

inline __device__ float3 signf(const float3 &v) {
    return make_float3(signf(v.x), signf(v.y), signf(v.z));
}

////////////////////////////////////////////////////
/////////////           utils          /////////////
////////////////////////////////////////////////////

// rays_o/d: [N, 3]
// nears/fars: [N]
// scalar_t should always be float in use.
template <typename scalar_t>
__global__ void kernel_near_far_from_aabb(
    const scalar_t * __restrict__ rays_o,
    const scalar_t * __restrict__ rays_d,
    const scalar_t * __restrict__ aabb,
    const uint32_t N,
    const float min_near,
    scalar_t * nears, scalar_t * fars
) {
    // parallel per ray
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= N) return;

    // locate
    rays_o += n * 3;
    rays_d += n * 3;

    const float ox = rays_o[0], oy = rays_o[1], oz = rays_o[2];
    const float dx = rays_d[0], dy = rays_d[1], dz = rays_d[2];
    const float rdx = 1 / dx, rdy = 1 / dy, rdz = 1 / dz;

    // get near far (assume cube scene)
    float near = (aabb[0] - ox) * rdx;
    float far = (aabb[3] - ox) * rdx;
    if (near > far) swapf(near, far);

    float near_y = (aabb[1] - oy) * rdy;
    float far_y = (aabb[4] - oy) * rdy;
    if (near_y > far_y) swapf(near_y, far_y);

    if (near > far_y || near_y > far) {
        nears[n] = fars[n] = std::numeric_limits<scalar_t>::max();
        return;
    }

    if (near_y > near) near = near_y;
    if (far_y < far) far = far_y;

    float near_z = (aabb[2] - oz) * rdz;
    float far_z = (aabb[5] - oz) * rdz;
    if (near_z > far_z) swapf(near_z, far_z);

    if (near > far_z || near_z > far) {
        nears[n] = fars[n] = std::numeric_limits<scalar_t>::max();
        return;
    }

    if (near_z > near) near = near_z;
    if (far_z < far) far = far_z;

    if (near < min_near) near = min_near;

    nears[n] = near;
    fars[n] = far;
}


void near_far_from_aabb(const at::Tensor rays_o, const at::Tensor rays_d, const at::Tensor aabb, const uint32_t N, const float min_near, at::Tensor nears, at::Tensor fars) {

    static constexpr uint32_t N_THREAD = 128;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    rays_o.scalar_type(), "near_far_from_aabb", ([&] {
        kernel_near_far_from_aabb<<<div_round_up(N, N_THREAD), N_THREAD>>>(rays_o.data_ptr<scalar_t>(), rays_d.data_ptr<scalar_t>(), aabb.data_ptr<scalar_t>(), N, min_near, nears.data_ptr<scalar_t>(), fars.data_ptr<scalar_t>());
    }));
}


// rays_o/d: [N, 3]
// radius: float
// coords: [N, 2]
template <typename scalar_t>
__global__ void kernel_sph_from_ray(
    const scalar_t * __restrict__ rays_o,
    const scalar_t * __restrict__ rays_d,
    const float radius,
    const uint32_t N,
    scalar_t * coords
) {
    // parallel per ray
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= N) return;

    // locate
    rays_o += n * 3;
    rays_d += n * 3;
    coords += n * 2;

    const float ox = rays_o[0], oy = rays_o[1], oz = rays_o[2];
    const float dx = rays_d[0], dy = rays_d[1], dz = rays_d[2];
    const float rdx = 1 / dx, rdy = 1 / dy, rdz = 1 / dz;

    // solve t from || o + td || = radius
    const float A = dx * dx + dy * dy + dz * dz;
    const float B = ox * dx + oy * dy + oz * dz; // in fact B / 2
    const float C = ox * ox + oy * oy + oz * oz - radius * radius;

    const float t = (- B + sqrtf(B * B - A * C)) / A; // always use the larger solution (positive)

    // solve theta, phi (assume y is the up axis)
    const float x = ox + t * dx, y = oy + t * dy, z = oz + t * dz;
    const float theta = atan2(sqrtf(x * x + z * z), y); // [0, PI)
    const float phi = atan2(z, x); // [-PI, PI)

    // normalize to [-1, 1]
    coords[0] = 2 * theta * RPI() - 1;
    coords[1] = phi * RPI();
}


void sph_from_ray(const at::Tensor rays_o, const at::Tensor rays_d, const float radius, const uint32_t N, at::Tensor coords) {

    static constexpr uint32_t N_THREAD = 128;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    rays_o.scalar_type(), "sph_from_ray", ([&] {
        kernel_sph_from_ray<<<div_round_up(N, N_THREAD), N_THREAD>>>(rays_o.data_ptr<scalar_t>(), rays_d.data_ptr<scalar_t>(), radius, N, coords.data_ptr<scalar_t>());
    }));
}


// coords: int32, [N, 3]
// indices: int32, [N]
__global__ void kernel_morton3D(
    const int * __restrict__ coords,
    const uint32_t N,
    int * indices
) {
    // parallel
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= N) return;

    // locate
    coords += n * 3;
    indices[n] = __morton3D(coords[0], coords[1], coords[2]);
}


void morton3D(const at::Tensor coords, const uint32_t N, at::Tensor indices) {
    static constexpr uint32_t N_THREAD = 128;
    kernel_morton3D<<<div_round_up(N, N_THREAD), N_THREAD>>>(coords.data_ptr<int>(), N, indices.data_ptr<int>());
}


// indices: int32, [N]
// coords: int32, [N, 3]
__global__ void kernel_morton3D_invert(
    const int * __restrict__ indices,
    const uint32_t N,
    int * coords
) {
    // parallel
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= N) return;

    // locate
    coords += n * 3;

    const int ind = indices[n];

    coords[0] = __morton3D_invert(ind >> 0);
    coords[1] = __morton3D_invert(ind >> 1);
    coords[2] = __morton3D_invert(ind >> 2);
}


void morton3D_invert(const at::Tensor indices, const uint32_t N, at::Tensor coords) {
    static constexpr uint32_t N_THREAD = 128;
    kernel_morton3D_invert<<<div_round_up(N, N_THREAD), N_THREAD>>>(indices.data_ptr<int>(), N, coords.data_ptr<int>());
}


// grid: float, [C, H, H, H]
// N: int, C * H * H * H / 8
// density_thresh: float
// bitfield: uint8, [N]
template <typename scalar_t>
__global__ void kernel_packbits(
    const scalar_t * __restrict__ grid,
    const uint32_t N,
    const float density_thresh,
    uint8_t * bitfield
) {
    // parallel per byte
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= N) return;

    // locate
    grid += n * 8;

    uint8_t bits = 0;

    #pragma unroll
    for (uint8_t i = 0; i < 8; i++) {
        bits |= (grid[i] > density_thresh) ? ((uint8_t)1 << i) : 0;
    }

    bitfield[n] = bits;
}


void packbits(const at::Tensor grid, const uint32_t N, const float density_thresh, at::Tensor bitfield) {

    static constexpr uint32_t N_THREAD = 128;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    grid.scalar_type(), "packbits", ([&] {
        kernel_packbits<<<div_round_up(N, N_THREAD), N_THREAD>>>(grid.data_ptr<scalar_t>(), N, density_thresh, bitfield.data_ptr<uint8_t>());
    }));
}

////////////////////////////////////////////////////
/////////////         training         /////////////
////////////////////////////////////////////////////

// rays_o/d: [N, 3]
// grid: [CHHH / 8]
// xyzs, dirs, deltas: [M, 3], [M, 3], [M, 2]
// dirs: [M, 3]
// rays: [N, 3], idx, offset, num_steps
template <typename scalar_t>
__global__ void kernel_march_rays_train(
    const scalar_t * __restrict__ rays_o,
    const scalar_t * __restrict__ rays_d,  
    const scalar_t * __restrict__ z_hats,
    const uint8_t * __restrict__ grid,
    const float bound,
    const float dt_gamma, const uint32_t max_steps, const bool is_ndc,
    const uint32_t N, const uint32_t C, const uint32_t H, const uint32_t M,
    const scalar_t* __restrict__ nears, 
    const scalar_t* __restrict__ fars,
    scalar_t * xyzs, scalar_t * dirs, scalar_t * deltas,
    int * rays,
    int * counter,
    const scalar_t* __restrict__ noises
) {
    // parallel per ray
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= N) return;

    // locate
    rays_o += n * 3;
    rays_d += n * 3;
    z_hats += n;

    // ray marching
    const float ox = rays_o[0], oy = rays_o[1], oz = rays_o[2];
    const float dx = rays_d[0], dy = rays_d[1], dz = rays_d[2];
    const float rdx = 1 / dx, rdy = 1 / dy, rdz = 1 / dz;
    const float rH = 1 / (float)H;
    const float H3 = H * H * H;

    const float near = nears[n];
    const float far = fars[n];
    const float noise = noises[n];

    const float dt_min = 2 * SQRT3() / max_steps;
    const float dt_max = 2 * SQRT3() * (1 << (C - 1)) / H;
    
    float t0 = near;
    
    // perturb
    t0 += clamp(t0 * dt_gamma, dt_min, dt_max) * noise;

    // first pass: estimation of num_steps
    float t = t0;
    uint32_t num_steps = 0;

    //if (t < far) printf("valid ray %d t=%f near=%f far=%f \n", n, t, near, far);
    
    while (t < far && num_steps < max_steps) {
        // current point
        const float x = clamp(ox + t * dx, -bound, bound);
        const float y = clamp(oy + t * dy, -bound, bound);
        const float z = clamp(oz + t * dz, -bound, bound);

        const float dt = clamp(t * dt_gamma, dt_min, dt_max);

        // get mip level
        const int level = max(mip_from_pos(x, y, z, C), mip_from_dt(dt, H, C)); // range in [0, C - 1]

        const float mip_bound = fminf(scalbnf(1.0f, level), bound);
        const float mip_rbound = 1 / mip_bound;
        
        // convert to nearest grid position
        const int nx = clamp(0.5 * (x * mip_rbound + 1) * H, 0.0f, (float)(H - 1));
        const int ny = clamp(0.5 * (y * mip_rbound + 1) * H, 0.0f, (float)(H - 1));
        const int nz = clamp(0.5 * (z * mip_rbound + 1) * H, 0.0f, (float)(H - 1));

        const uint32_t index = level * H3 + __morton3D(nx, ny, nz);
        const bool occ = grid[index / 8] & (1 << (index % 8));

        // if occpuied, advance a small step, and write to output
        //if (n == 0) printf("t=%f density=%f vs thresh=%f step=%d\n", t, density, density_thresh, num_steps);

        if (occ) {
            num_steps++;
            t += dt;
        // else, skip a large step (basically skip a voxel grid)
        } else {
            // calc distance to next voxel
            const float tx = (((nx + 0.5f + 0.5f * signf(dx)) * rH * 2 - 1) * mip_bound - x) * rdx;
            const float ty = (((ny + 0.5f + 0.5f * signf(dy)) * rH * 2 - 1) * mip_bound - y) * rdy;
            const float tz = (((nz + 0.5f + 0.5f * signf(dz)) * rH * 2 - 1) * mip_bound - z) * rdz;

            const float tt = t + fmaxf(0.0f, fminf(tx, fminf(ty, tz)));
            // step until next voxel
            do { 
                t += clamp(t * dt_gamma, dt_min, dt_max);
            } while (t < tt);
        }
    }

    //printf("[n=%d] num_steps=%d, near=%f, far=%f, dt=%f, max_steps=%f\n", n, num_steps, near, far, dt_min, (far - near) / dt_min);

    // second pass: really locate and write points & dirs
    uint32_t point_index = atomicAdd(counter, num_steps);
    uint32_t ray_index = atomicAdd(counter + 1, 1);
    
    //printf("[n=%d] num_steps=%d, point_index=%d, ray_index=%d\n", n, num_steps, point_index, ray_index);

    // write rays
    rays[ray_index * 3] = n;
    rays[ray_index * 3 + 1] = point_index;
    rays[ray_index * 3 + 2] = num_steps;

    if (num_steps == 0) return;
    if (point_index + num_steps >= M) return;

    xyzs += point_index * 3;
    dirs += point_index * 3;
    deltas += point_index * 4;

    t = t0;
    uint32_t step = 0;

    float last_t = t;
    float last_z = clamp(oz + t * dz, -bound, bound);
    float new_z;

    while (t < far && step < num_steps) {
        // current point
        const float x = clamp(ox + t * dx, -bound, bound);
        const float y = clamp(oy + t * dy, -bound, bound);
        const float z = clamp(oz + t * dz, -bound, bound);

        const float dt = clamp(t * dt_gamma, dt_min, dt_max);

        // get mip level
        const int level = max(mip_from_pos(x, y, z, C), mip_from_dt(dt, H, C)); // range in [0, C - 1]

        const float mip_bound = fminf(scalbnf(1.0f, level), bound);
        const float mip_rbound = 1 / mip_bound;
        
        // convert to nearest grid position
        const int nx = clamp(0.5 * (x * mip_rbound + 1) * H, 0.0f, (float)(H - 1));
        const int ny = clamp(0.5 * (y * mip_rbound + 1) * H, 0.0f, (float)(H - 1));
        const int nz = clamp(0.5 * (z * mip_rbound + 1) * H, 0.0f, (float)(H - 1));

        // query grid
        const uint32_t index = level * H3 + __morton3D(nx, ny, nz);
        const bool occ = grid[index / 8] & (1 << (index % 8));

        // if occpuied, advance a small step, and write to output
        if (occ) {
            // write step
            xyzs[0] = x;
            xyzs[1] = y;
            xyzs[2] = z;
            dirs[0] = dx;
            dirs[1] = dy;
            dirs[2] = dz;
            t += dt;
            deltas[0] = dt;
            deltas[1] = t - last_t; // used to calc depth
            last_t = t;
            if (is_ndc) {
                new_z = clamp(oz + t * dz, -bound, bound);
                deltas[2] = (2 / (new_z - 1) - 2 / (z - 1)) / z_hats[0];
                deltas[3] = (2 / (new_z - 1) - 2 / (last_z - 1)) / z_hats[0];
                last_z = z;
            }
            xyzs += 3;
            dirs += 3;
            deltas += 4;
            step++;
        // else, skip a large step (basically skip a voxel grid)
        } else {
            // calc distance to next voxel
            const float tx = (((nx + 0.5f + 0.5f * signf(dx)) * rH * 2 - 1) * mip_bound - x) * rdx;
            const float ty = (((ny + 0.5f + 0.5f * signf(dy)) * rH * 2 - 1) * mip_bound - y) * rdy;
            const float tz = (((nz + 0.5f + 0.5f * signf(dz)) * rH * 2 - 1) * mip_bound - z) * rdz;
            const float tt = t + fmaxf(0.0f, fminf(tx, fminf(ty, tz)));
            // step until next voxel
            do { 
                t += clamp(t * dt_gamma, dt_min, dt_max); 
            } while (t < tt);
        }
    }
}

void march_rays_train(const at::Tensor rays_o, const at::Tensor rays_d, const at::Tensor z_hats, const at::Tensor grid, const float bound, const float dt_gamma, const uint32_t max_steps, const bool is_ndc, const uint32_t N, const uint32_t C, const uint32_t H, const uint32_t M, const at::Tensor nears, const at::Tensor fars, at::Tensor xyzs, at::Tensor dirs, at::Tensor deltas, at::Tensor rays, at::Tensor counter, at::Tensor noises) {

    static constexpr uint32_t N_THREAD = 128;
    
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    rays_o.scalar_type(), "march_rays_train", ([&] {
        kernel_march_rays_train<<<div_round_up(N, N_THREAD), N_THREAD>>>(rays_o.data_ptr<scalar_t>(), rays_d.data_ptr<scalar_t>(), z_hats.data_ptr<scalar_t>(), grid.data_ptr<uint8_t>(), bound, dt_gamma, max_steps, is_ndc, N, C, H, M, nears.data_ptr<scalar_t>(), fars.data_ptr<scalar_t>(), xyzs.data_ptr<scalar_t>(), dirs.data_ptr<scalar_t>(), deltas.data_ptr<scalar_t>(), rays.data_ptr<int>(), counter.data_ptr<int>(), noises.data_ptr<scalar_t>());
    }));
}


template <typename scalar_t>
__global__ void kernel_march_rays_unbounded_train(
    const scalar_t * __restrict__ rays_o,
    const scalar_t * __restrict__ rays_d,
    const uint8_t * __restrict__ grid,
    const float bound,
    const float dt_gamma,
    const uint32_t max_steps,
    const uint32_t N,
    const uint32_t C,
    const uint32_t H,
    const uint32_t M,
    const scalar_t * __restrict__ nears,
    const scalar_t * __restrict__ fars,
    scalar_t * xyzs,
    scalar_t * dirs,
    scalar_t * deltas,
    int * rays,
    int * counter,
    const scalar_t * __restrict__ noises
) {
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= N)
        return;
    
    rays_o += n * 3;
    rays_d += n * 3;

    const float3 o = make_float3(rays_o[0], rays_o[1], rays_o[2]);
    const float3 d = make_float3(rays_d[0], rays_d[1], rays_d[2]);
    const float3 rd = 1 / d;

    const float rH = 1 / (float)H;
    const float H3 = H * H * H;

    const float near = nears[n];
    const float far = fars[n];
    const float noise = noises[n];

    const float dt_min = 2 * SQRT3() / max_steps;
    const float dt_max = 2 * SQRT3() * (1 << (C - 1)) / H;

    float t0 = near;
    t0 += clamp(t0 * dt_gamma, dt_min, dt_max) * noise;

    float t = t0;
    uint32_t num_steps = 0;

    while (t < far && num_steps < max_steps) {
        const float3 p = clamp(o + t * d, -bound, bound);
        const float dt = clamp(t * dt_gamma, dt_min, dt_max);

        const int level = max(mip_from_pos(p, C), mip_from_dt(dt, H, C)); // [0, C)
        const float mip_bound = fminf(scalbnf(1.0f, level), bound); // min(2^lvl, bound)
        const float mip_rbound = 1 / mip_bound;

        // corresponding cell for p
        const uint3 np = to_uint3(clamp(0.5 * (p * mip_rbound + 1) * H, 0.0f, (float)(H - 1)));
        const uint32_t index = level * H3 + __morton3D(np);
        const bool occ = grid[index / 8] & (1 << (index % 8));

        if (occ) {
            num_steps++;
            t += dt;
        } else {
            const float3 next_ts = ((
                (to_float3(np) + 0.5f + 0.5f * signf(d)) * rH * 2 - 1) * mip_bound - p) * rd;
            const float tt = t + fmaxf(0.0f, fminf(next_ts));

            do {
                t += clamp(t * dt_gamma, dt_min, dt_max);
            } while (t < tt);
        }
    }

    float r0 = 1 / norm(clamp(o + t * d, -bound, bound));
    float r = r0;
    float thres_r = 0.05;  // closer to 0 -> render into infinity
    float dr = 0.005;

    const float tb = -dot(d, o);  // assuming |d| = 1
    const float b = norm(o + tb * d);  // B = closest pt on ray to origin; refer to NeRF++ paper

    while (r > thres_r) {
        r -= dr;
        const float tr = sqrtf(1 / (r * r) - b * b) + tb;
        const float3 p = o + tr * d;  // point in Euclidean space
        const float3 pinv = p * r * r;  // point in inverted sphere space

        const uint3 np = to_uint3(clamp(0.5 * (p + 1) * H, 0.0f, (float)(H - 1)));
        const uint32_t index = C * H3 + __morton3D(np);
        const bool occ = grid[index / 8] & (1 << (index % 8));
        
        if (occ) {
            num_steps++;
        }
    }

    uint32_t point_index = atomicAdd(counter, num_steps);
    uint32_t ray_index = atomicAdd(counter + 1, 1);

    rays[ray_index * 3] = n;
    rays[ray_index * 3 + 1] = point_index;
    rays[ray_index * 3 + 2] = num_steps;

    return;

    if ((num_steps == 0) || (point_index + num_steps >= M))
        return;

    xyzs += point_index * 3;
    dirs += point_index * 3;
    deltas += point_index * 4;

    t = t0;
    uint32_t step = 0;
    float last_t = t;

    while (t < far && step < num_steps) {
        const float3 p = clamp(o + t * d, -bound, bound);
        const float dt = clamp(t * dt_gamma, dt_min, dt_max);

        const int level = max(mip_from_pos(p, C), mip_from_dt(dt, H, C)); // [0, C)
        const float mip_bound = fminf(scalbnf(1.0f, level), bound); // min(2^lvl, bound)
        const float mip_rbound = 1 / mip_bound;

        // corresponding cell for p
        const uint3 np = to_uint3(clamp(0.5 * (p * mip_rbound + 1) * H, 0.0f, (float)(H - 1)));
        const uint32_t index = level * H3 + __morton3D(np);
        const bool occ = grid[index / 8] & (1 << (index % 8));

        if (occ) {
            write_float3(p, xyzs);
            write_float3(d, dirs);
            t += dt;
            deltas[0] = dt;
            deltas[1] = t - last_t;
            last_t = t;
            xyzs += 3;
            dirs += 3;
            step++;
        } else {
            const float3 next_ts = ((
                (to_float3(np) + 0.5f + 0.5f * signf(d)) * rH * 2 - 1) * mip_bound - p) * rd;
            const float tt = t + fmaxf(0.0f, fminf(next_ts));

            do {
                t += clamp(t * dt_gamma, dt_min, dt_max);
            } while (t < tt);
        }
    }
}

void march_rays_unbounded_train(
    const at::Tensor rays_o,
    const at::Tensor rays_d,
    const at::Tensor grid,
    const float bound,
    const float dt_gamma,
    const uint32_t max_steps,
    const uint32_t N,
    const uint32_t C,
    const uint32_t H,
    const uint32_t M,
    const at::Tensor nears,
    const at::Tensor fars,
    at::Tensor xyzs,
    at::Tensor dirs,
    at::Tensor deltas,
    at::Tensor rays,
    at::Tensor counter,
    at::Tensor noises
) {
    static constexpr uint32_t N_THREAD = 128;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
        rays_o.scalar_type(),
        "march_rays_train",
        ([&] {
            kernel_march_rays_unbounded_train<<<div_round_up(N, N_THREAD), N_THREAD>>>(
                rays_o.data_ptr<scalar_t>(),
                rays_d.data_ptr<scalar_t>(),
                grid.data_ptr<uint8_t>(),
                bound, dt_gamma, max_steps, N, C, H, M,
                nears.data_ptr<scalar_t>(),
                fars.data_ptr<scalar_t>(),
                xyzs.data_ptr<scalar_t>(),
                dirs.data_ptr<scalar_t>(),
                deltas.data_ptr<scalar_t>(),
                rays.data_ptr<int>(),
                counter.data_ptr<int>(),
                noises.data_ptr<scalar_t>()
            );
        })
    );
}

// sigmas: [M]
// rgbs: [M, C]
// deltas: [M, 2]
// rays: [N, 3], idx, offset, num_steps
// weights_sum: [N], final pixel alpha
// depth: [N,]
// image: [N, C]
template <typename scalar_t>
__global__ void kernel_composite_rays_train_forward(
    const scalar_t * __restrict__ sigmas,
    const scalar_t * __restrict__ rgbs,  
    const scalar_t * __restrict__ deltas,
    const int * __restrict__ rays,
    const uint32_t M, const uint32_t N, const uint32_t C,
    const float T_thresh, const bool is_ndc,
    scalar_t * weights_sum,
    scalar_t * depth,
    scalar_t * image
) {
    // parallel per ray
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= N) return;

    // locate 
    uint32_t index = rays[n * 3];
    uint32_t offset = rays[n * 3 + 1];
    uint32_t num_steps = rays[n * 3 + 2];

    // empty ray, or ray that exceed max step count.
    for (int i = 0; i < C; ++i)
        image[index * C + i] = 0;
    if (num_steps == 0 || offset + num_steps >= M) {
        weights_sum[index] = 0;
        depth[index] = 0;
        return;
    }

    sigmas += offset;
    rgbs += offset * C;
    deltas += offset * 4;

    // accumulate 
    uint32_t step = 0;

    scalar_t T = 1.0f;
    scalar_t ws = 0, t = 0, d = 0;

    while (step < num_steps) {

        const scalar_t alpha = 1.0f - __expf(- sigmas[0] * (is_ndc ? deltas[2] : deltas[0]));
        const scalar_t weight = alpha * T;

        for (int i = 0; i < C; ++i)
            image[index * C + i] += weight * rgbs[i];
        
        t += (is_ndc ? deltas[3] : deltas[1]); // real delta
        d += weight * t;
        
        ws += weight;
        
        T *= 1.0f - alpha;

        // minimal remained transmittence
        if (T < T_thresh) break;

        //printf("[n=%d] num_steps=%d, alpha=%f, w=%f, T=%f, sum_dt=%f, d=%f\n", n, step, alpha, weight, T, sum_delta, d);

        // locate
        sigmas++;
        rgbs += C;
        deltas += 4;

        step++;
    }

    //printf("[n=%d] rgb=(%f, %f, %f), d=%f\n", n, r, g, b, d);

    // write
    weights_sum[index] = ws; // weights_sum
    depth[index] = d;
}


void composite_rays_train_forward(const at::Tensor sigmas, const at::Tensor rgbs, const at::Tensor deltas, const at::Tensor rays, const uint32_t M, const uint32_t N, const uint32_t C, const float T_thresh, const bool is_ndc, at::Tensor weights_sum, at::Tensor depth, at::Tensor image) {

    static constexpr uint32_t N_THREAD = 128;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    sigmas.scalar_type(), "composite_rays_train_forward", ([&] {
        kernel_composite_rays_train_forward<<<div_round_up(N, N_THREAD), N_THREAD>>>(sigmas.data_ptr<scalar_t>(), rgbs.data_ptr<scalar_t>(), deltas.data_ptr<scalar_t>(), rays.data_ptr<int>(), M, N, C, T_thresh, is_ndc, weights_sum.data_ptr<scalar_t>(), depth.data_ptr<scalar_t>(), image.data_ptr<scalar_t>());
    }));
}


// grad_weights_sum: [N,]
// grad: [N, C]
// sigmas: [M]
// rgbs: [M, C]
// deltas: [M, 2]
// rays: [N, 3], idx, offset, num_steps
// weights_sum: [N,], weights_sum here 
// image: [N, C]
// grad_sigmas: [M]
// grad_rgbs: [M, C]
// rgbs_buf: [N, C]
template <typename scalar_t>
__global__ void kernel_composite_rays_train_backward(
    const scalar_t * __restrict__ grad_weights_sum,
    const scalar_t * __restrict__ grad_image,
    const scalar_t * __restrict__ sigmas,
    const scalar_t * __restrict__ rgbs, 
    const scalar_t * __restrict__ deltas,
    const int * __restrict__ rays,
    const bool is_ndc,
    const scalar_t * __restrict__ weights_sum,
    const scalar_t * __restrict__ image,
    const uint32_t M, const uint32_t N, const uint32_t C, const float T_thresh,
    scalar_t * grad_sigmas,
    scalar_t * grad_rgbs,
    scalar_t * rgbs_buf
) {
    // parallel per ray
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= N) return;

    // locate 
    uint32_t index = rays[n * 3];
    uint32_t offset = rays[n * 3 + 1];
    uint32_t num_steps = rays[n * 3 + 2];

    if (num_steps == 0 || offset + num_steps >= M) return;

    grad_weights_sum += index;
    grad_image += index * C;
    weights_sum += index;
    image += index * C;
    sigmas += offset;
    rgbs += offset * C;
    deltas += offset * 4;
    grad_sigmas += offset;
    grad_rgbs += offset * C;
    rgbs_buf += index * C;

    // accumulate 
    uint32_t step = 0;
    
    scalar_t T = 1.0f;
    const scalar_t ws_final = weights_sum[0];
    scalar_t ws = 0;

    while (step < num_steps) {
        
        const scalar_t alpha = 1.0f - __expf(- sigmas[0] * (is_ndc ? deltas[2] : deltas[0]));
        const scalar_t weight = alpha * T;

        for (int i = 0; i < C; ++i)
            rgbs_buf[i] += weight * rgbs[i];
        ws += weight;

        T *= 1.0f - alpha;

        // minimal remained transmittence
        if (T < T_thresh) break;

        // check https://note.kiui.moe/others/nerf_gradient/ for the gradient calculation.
        // write grad_rgbs
        for (int i = 0; i < C; ++i)
            grad_rgbs[i] = grad_image[i] * weight;

        // write grad_sigmas
        scalar_t grad_image_sum = 0;
        for (int i = 0; i < C; ++i)
            grad_image_sum += (grad_image[i] * (T * rgbs[i] - (image[i] - rgbs_buf[i])));
        grad_sigmas[0] = (is_ndc ? deltas[2] : deltas[0]) * (
            grad_image_sum + grad_weights_sum[0] * (1 - ws_final));

        //printf("[n=%d] num_steps=%d, T=%f, grad_sigmas=%f, r_final=%f, r=%f\n", n, step, T, grad_sigmas[0], r_final, r);
    
        // locate
        sigmas++;
        rgbs += C;
        deltas += 4;
        grad_sigmas++;
        grad_rgbs += C;

        step++;
    }
}


void composite_rays_train_backward(const at::Tensor grad_weights_sum, const at::Tensor grad_image, const at::Tensor sigmas, const at::Tensor rgbs, const at::Tensor deltas, const at::Tensor rays, const bool is_ndc, const at::Tensor weights_sum, const at::Tensor image, const uint32_t M, const uint32_t N, const uint32_t C, const float T_thresh, at::Tensor grad_sigmas, at::Tensor grad_rgbs, at::Tensor rgbs_buf) {

    static constexpr uint32_t N_THREAD = 128;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    grad_image.scalar_type(), "composite_rays_train_backward", ([&] {
        kernel_composite_rays_train_backward<<<div_round_up(N, N_THREAD), N_THREAD>>>(grad_weights_sum.data_ptr<scalar_t>(), grad_image.data_ptr<scalar_t>(), sigmas.data_ptr<scalar_t>(), rgbs.data_ptr<scalar_t>(), deltas.data_ptr<scalar_t>(), rays.data_ptr<int>(), is_ndc, weights_sum.data_ptr<scalar_t>(), image.data_ptr<scalar_t>(), M, N, C, T_thresh, grad_sigmas.data_ptr<scalar_t>(), grad_rgbs.data_ptr<scalar_t>(), rgbs_buf.data_ptr<scalar_t>());
    }));
}


////////////////////////////////////////////////////
/////////////          infernce        /////////////
////////////////////////////////////////////////////

template <typename scalar_t>
__global__ void kernel_march_rays(
    const uint32_t n_alive, 
    const uint32_t n_step, 
    const int* __restrict__ rays_alive, 
    const scalar_t* __restrict__ rays_t, 
    const scalar_t* __restrict__ rays_o, 
    const scalar_t* __restrict__ rays_d, 
    const scalar_t* __restrict__ z_hats,
    const float bound,
    const float dt_gamma, const uint32_t max_steps, const bool is_ndc,
    const uint32_t C, const uint32_t H,
    const uint8_t * __restrict__ grid,
    const scalar_t* __restrict__ nears,
    const scalar_t* __restrict__ fars,
    scalar_t* xyzs, scalar_t* dirs, scalar_t* deltas,
    const scalar_t* __restrict__ noises
) {
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= n_alive) return;

    const int index = rays_alive[n]; // ray id
    const float noise = noises[n];
    
    // locate
    rays_o += index * 3;
    rays_d += index * 3;
    rays_t += index * (is_ndc ? 2 : 1);
    z_hats += index;
    xyzs += n * n_step * 3;
    dirs += n * n_step * 3;
    deltas += n * n_step * 4;
    
    const float ox = rays_o[0], oy = rays_o[1], oz = rays_o[2];
    const float dx = rays_d[0], dy = rays_d[1], dz = rays_d[2];
    const float rdx = 1 / dx, rdy = 1 / dy, rdz = 1 / dz;
    const float rH = 1 / (float)H;
    const float H3 = H * H * H;
    
    float t = rays_t[0]; // current ray's t
    const float near = nears[index], far = fars[index];

    const float dt_min = 2 * SQRT3() / max_steps;
    const float dt_max = 2 * SQRT3() * (1 << (C - 1)) / H;

    // march for n_step steps, record points
    uint32_t step = 0;

    // introduce some randomness
    t += clamp(t * dt_gamma, dt_min, dt_max) * noise;

    float last_t = t;
    float last_z = clamp(oz + t * dz, -bound, bound);
    float new_z;

    while (t < far && step < n_step) {
        // current point
        const float x = clamp(ox + t * dx, -bound, bound);
        const float y = clamp(oy + t * dy, -bound, bound);
        const float z = clamp(oz + t * dz, -bound, bound);

        const float dt = clamp(t * dt_gamma, dt_min, dt_max);

        // get mip level
        const int level = max(mip_from_pos(x, y, z, C), mip_from_dt(dt, H, C)); // range in [0, C - 1]

        const float mip_bound = fminf(scalbnf(1, level), bound);
        const float mip_rbound = 1 / mip_bound;
        
        // convert to nearest grid position
        const int nx = clamp(0.5 * (x * mip_rbound + 1) * H, 0.0f, (float)(H - 1));
        const int ny = clamp(0.5 * (y * mip_rbound + 1) * H, 0.0f, (float)(H - 1));
        const int nz = clamp(0.5 * (z * mip_rbound + 1) * H, 0.0f, (float)(H - 1));

        const uint32_t index = level * H3 + __morton3D(nx, ny, nz);
        const bool occ = grid[index / 8] & (1 << (index % 8));

        // if occpuied, advance a small step, and write to output
        if (occ) {
            // write step
            xyzs[0] = x;
            xyzs[1] = y;
            xyzs[2] = z;
            dirs[0] = dx;
            dirs[1] = dy;
            dirs[2] = dz;
            // calc dt
            t += dt;
            deltas[0] = dt;
            deltas[1] = t - last_t; // used to calc depth
            if (is_ndc) {
                new_z = clamp(oz + t * dz, -bound, bound);
                deltas[2] = (2 / (new_z - 1) - 2 / (z - 1)) / z_hats[0];
                deltas[3] = (2 / (new_z - 1) - 2 / (last_z - 1)) / z_hats[0];
                last_z = new_z;
            }
            last_t = t;
            // step
            xyzs += 3;
            dirs += 3;
            deltas += 4;
            step++;

        // else, skip a large step (basically skip a voxel grid)
        } else {
            // calc distance to next voxel
            const float tx = (((nx + 0.5f + 0.5f * signf(dx)) * rH * 2 - 1) * mip_bound - x) * rdx;
            const float ty = (((ny + 0.5f + 0.5f * signf(dy)) * rH * 2 - 1) * mip_bound - y) * rdy;
            const float tz = (((nz + 0.5f + 0.5f * signf(dz)) * rH * 2 - 1) * mip_bound - z) * rdz;
            const float tt = t + fmaxf(0.0f, fminf(tx, fminf(ty, tz)));
            // step until next voxel
            do { 
                t += clamp(t * dt_gamma, dt_min, dt_max);
            } while (t < tt);
        }
    }
}


void march_rays(const uint32_t n_alive, const uint32_t n_step, const at::Tensor rays_alive, const at::Tensor rays_t, const at::Tensor rays_o, const at::Tensor rays_d, const at::Tensor z_hats, const float bound, const float dt_gamma, const uint32_t max_steps, const bool is_ndc, const uint32_t C, const uint32_t H, const at::Tensor grid, const at::Tensor near, const at::Tensor far, at::Tensor xyzs, at::Tensor dirs, at::Tensor deltas, at::Tensor noises) {
    static constexpr uint32_t N_THREAD = 128;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    rays_o.scalar_type(), "march_rays", ([&] {
        kernel_march_rays<<<div_round_up(n_alive, N_THREAD), N_THREAD>>>(n_alive, n_step, rays_alive.data_ptr<int>(), rays_t.data_ptr<scalar_t>(), rays_o.data_ptr<scalar_t>(), rays_d.data_ptr<scalar_t>(), z_hats.data_ptr<scalar_t>(), bound, dt_gamma, max_steps, is_ndc, C, H, grid.data_ptr<uint8_t>(), near.data_ptr<scalar_t>(), far.data_ptr<scalar_t>(), xyzs.data_ptr<scalar_t>(), dirs.data_ptr<scalar_t>(), deltas.data_ptr<scalar_t>(), noises.data_ptr<scalar_t>());
    }));
}


template <typename scalar_t>
__global__ void kernel_composite_rays(
    const uint32_t n_alive, 
    const uint32_t n_step, 
    const float T_thresh,
    int* rays_alive, 
    scalar_t* rays_t, 
    const scalar_t* __restrict__ sigmas, 
    const scalar_t* __restrict__ rgbs, 
    const scalar_t* __restrict__ deltas, 
    const uint32_t C,
    const bool is_ndc,
    scalar_t* weights_sum, scalar_t* depth, scalar_t* image
) {
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= n_alive) return;

    const int index = rays_alive[n]; // ray id
    
    // locate 
    sigmas += n * n_step;
    rgbs += n * n_step * C;
    deltas += n * n_step * 4;
    
    rays_t += index * (is_ndc ? 2 : 1);
    weights_sum += index;
    depth += index;
    image += index * C;

    scalar_t t_rm, t_phy;
    if (is_ndc) {
        t_rm = rays_t[0];
        t_phy = rays_t[1];
    } else {
        t_phy = rays_t[0];
    }
    
    scalar_t weight_sum = weights_sum[0];
    scalar_t d = depth[0];

    // accumulate 
    uint32_t step = 0;
    while (step < n_step) {
        
        // ray is terminated if delta == 0
        if (deltas[0] == 0) break;
        
        const scalar_t alpha = 1.0f - __expf(- sigmas[0] * (is_ndc ? deltas[2] : deltas[0]));

        /* 
        T_0 = 1; T_i = \prod_{j=0}^{i-1} (1 - alpha_j)
        w_i = alpha_i * T_i
        --> 
        T_i = 1 - \sum_{j=0}^{i-1} w_j
        */
        const scalar_t T = 1 - weight_sum;
        const scalar_t weight = alpha * T;
        weight_sum += weight;

        if (is_ndc) {
            t_rm += deltas[1];
            t_phy += deltas[3];
        } else {
            t_phy += deltas[1];
        }
        d += weight * t_phy;
        for (int i = 0; i < C; ++i)
            image[i] += weight * rgbs[i];

        //printf("[n=%d] num_steps=%d, alpha=%f, w=%f, T=%f, sum_dt=%f, d=%f\n", n, step, alpha, weight, T, sum_delta, d);

        // ray is terminated if T is too small
        // use a larger bound to further accelerate inference
        if (T < T_thresh) break;

        // locate
        sigmas++;
        rgbs += C;
        deltas += 4;
        step++;
    }

    //printf("[n=%d] rgb=(%f, %f, %f), d=%f\n", n, r, g, b, d);

    // rays_alive = -1 means ray is terminated early.
    if (step < n_step) {
        rays_alive[n] = -1;
    } else {
        if (is_ndc) {
            rays_t[0] = t_rm;
            rays_t[1] = t_phy;
        } else {
            rays_t[0] = t_phy;
        }
    }

    weights_sum[0] = weight_sum; // this is the thing I needed!
    depth[0] = d;
}


void composite_rays(const uint32_t n_alive, const uint32_t n_step, const float T_thresh, at::Tensor rays_alive, at::Tensor rays_t, const at::Tensor sigmas, const at::Tensor rgbs, const at::Tensor deltas, const uint32_t C, const bool is_ndc, at::Tensor weights, at::Tensor depth, at::Tensor image) {
    static constexpr uint32_t N_THREAD = 128;
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    image.scalar_type(), "composite_rays", ([&] {
        kernel_composite_rays<<<div_round_up(n_alive, N_THREAD), N_THREAD>>>(n_alive, n_step, T_thresh, rays_alive.data_ptr<int>(), rays_t.data_ptr<scalar_t>(), sigmas.data_ptr<scalar_t>(), rgbs.data_ptr<scalar_t>(), deltas.data_ptr<scalar_t>(), C, is_ndc, weights.data_ptr<scalar_t>(), depth.data_ptr<scalar_t>(), image.data_ptr<scalar_t>());
    }));
}